#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

//Parallel (GPU) function for mean filter
__global__ void meanFilter(int* imageArray, int* filteredArray, int img_height, int img_width, int window_width){
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
 
    if(x >= window_width/2 && x< (img_width- window_width/2) && y >= window_width/2 && y<(img_height-window_width/2)){
       int sum = 0;
          for(int wy = 0 - window_width/2; wy <= window_width/2 ; wy++){
             for(int wx = 0 - window_width/2; wx <= window_width/2 ; wx++){
                sum += imageArray[img_width*(y+wy) + (x+wx)];
             }
          }
          filteredArray[img_width*y + x] = sum / (window_width*window_width);
    }   
 }


int* getImageArrayFromBMP(FILE* fptr, int* height, int* width){
    int * imageArray;
    int offset;
    long n;

    fseek(fptr, 10, SEEK_SET);
    fread(&offset, 1, 4, fptr);
    fseek(fptr, 4, SEEK_CUR);
    fread(height, 1, 4, fptr);
    fread(width, 1, 4, fptr);

    imageArray = (int *) malloc((*height)*(*width)*sizeof(int));
    fseek(fptr, offset, SEEK_SET);
    for(int i=0; i < (*height)*(*width); i++){
       n = fread(&imageArray[i], 1, 1, fptr);
       if (n!=1) {
          printf("File not found");
      }
    }
    return imageArray;
}

//sequential (CPU) function for mean filter
void meanFilter_h(int* sourceArray, int* filteredArray, int height, int width, int window_width){

    for(int y = window_width/2; y < height - (window_width/2); y++){
       for(int x = window_width/2; x < width - (window_width/2); x++){
          int sum = 0;
          for(int wy = 0 - window_width/2; wy <= window_width/2 ; wy++){
             for(int wx = 0 - window_width/2; wx <= window_width/2 ; wx++){
                sum += sourceArray[width*(y+wy) + (x+wx)];
             }
          }
          filteredArray[width*y + x] = sum / (window_width*window_width);
       }
    }
}

int main(int argc,char **argv){

    int *sourceImage, *filteredImage;
    int height,width;
    int window_width = 5;

    printf("Mean filter program\n");
    FILE *fptr;
    fptr = fopen("puppy_1280.bmp", "r");
    if(fptr == NULL)
    {
       printf("Error!");   
       exit(1);             
    }

    sourceImage = getImageArrayFromBMP(fptr, &height, &width);
    fclose(fptr);

    filteredImage = (int *) malloc((height)*(width)*sizeof(int));

    clock_t start_h=clock();
    meanFilter_h(sourceImage, filteredImage, height, width, window_width);
    clock_t end_h = clock();
    double time_h = (double)(end_h - start_h)/CLOCKS_PER_SEC;

    int* d_image;
    int* d_filteredImage;
    int* h_filteredImage;

    h_filteredImage = (int *) malloc(height*width*sizeof(int));
    for(int i = 0; i< height*width; i++){
        h_filteredImage[i] = 0;
    }  

    hipMalloc((void **)&d_image, height*width*sizeof(int));    
    hipMalloc((void **)&d_filteredImage, height*width*sizeof(int));
    hipMemcpy(d_image, sourceImage, height*width*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32,32);
    dim3 numBlocks(1 + ((width-1)/threadsPerBlock.x), 1 + ((height-1)/threadsPerBlock.y));

    clock_t start_d=clock();
    meanFilter<<<numBlocks, threadsPerBlock>>>(d_image, d_filteredImage, height, width, window_width);
    hipDeviceSynchronize();
    clock_t end_d = clock();
    double time_d = (double)(end_d - start_d)/CLOCKS_PER_SEC;


    hipMemcpy(h_filteredImage, d_filteredImage, height*width*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_image);
    hipFree(d_filteredImage);

    printf("For %dx%d image and window size %d, CPU time %f is GPU time %f\n", height, width, window_width, time_h, time_d);

    free(filteredImage);
    free(sourceImage);
    free(h_filteredImage);    
    return 0;
}