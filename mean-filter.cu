#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

int** getArrayFromBMP(FILE* fptr){
   int** imageArray;
   int height, width, offset;
   long n;

   fseek(fptr, 10, SEEK_SET);
   fread(&offset, 1, 4, fptr);
   fseek(fptr, 4, SEEK_CUR);
   fread(&height, 1, 4, fptr);
   fread(&width, 1, 4, fptr);

   imageArray = (int**) malloc(height*sizeof(int*));
   for (int i=0; i<height; i++){
      imageArray[i] = (int*) malloc(width*sizeof(int));
   }

   fseek(fptr, offset, SEEK_SET);
   for (int y=height-1; y>=0; y--) {
      for (int x=0; x<width; x++) {
          n=fread(&imageArray[y][x], 1, 1, fptr);
          if (n!=1) {
              printf("File not found");
          }
      }
   }

   for(int i=0; i<height; i++){
      for(int j=0; j<width; j++){
         printf("%d ", imageArray[i][j]);
      }
      printf("\n");
   }

   return imageArray;
}

int main(int argc,char **argv)
{
   printf("Mean filter program\n");
   FILE *fptr;
   fptr = fopen("pup.bmp", "r");
   if(fptr == NULL)
   {
      printf("Error!");   
      exit(1);             
   }

   int** imageArray = getArrayFromBMP(fptr);

   fclose(fptr);
   printf("end\n");
   return 0;
}